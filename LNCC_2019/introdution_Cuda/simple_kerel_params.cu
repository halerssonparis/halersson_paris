
#include <hip/hip_runtime.h>
#include <stdio.h>
// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.


inline
hipError_t checkCuda(hipError_t result)
{
	#if defined(DEBUG) || defined(_DEBUG)
	if (result != cudaSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
		assert(result == cudaSuccess);
	}
	#endif
		return result;
}


__global__ void add( int a, int b, int *c ) {
	*c = a + b;
}


int main( void ) {
	int c;
	int *dev_c;
	checkCuda( hipMalloc( (void**)&dev_c, sizeof(int) ) );
	add<<<1,1>>>( 2, 7, dev_c );

	checkCuda( hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost ) );

	printf( "2 + 7 = %d\n", c );
	checkCuda( hipFree( dev_c ) );
	return 0;
}
